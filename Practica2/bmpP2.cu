#include "hip/hip_runtime.h"
// Joan Benlloch García y Gonzalo de Antonio Sierra

#include "bmpP2.h"

#define ANCHO_IMAGEN 800

template<typename T>
class gpuDataManager {
    public:
        T* gpuData;
        T* cpuData;
        int size;
        
    gpuDataManager(int size){
        hipMalloc(&gpuData, sizeof(T)*size);
        cpuData = new T[size];
        this->size = size;
    }
    
    void copyToGPU(){
        hipMemcpy(gpuData, cpuData, sizeof(T)*size, hipMemcpyHostToDevice);
    }
    
    void copyToCPU(){
        hipMemcpy(cpuData, gpuData, sizeof(T)*size, hipMemcpyDeviceToHost);
    }
    
    ~gpuDataManager(){
        hipFree(gpuData);
        delete[] cpuData;
    }
};

__global__ void applyBilinearFilter_k(int width, int height, int filter, float *data, float *dataOut)
{
    float filters[6][3][3] ={
        {
            // Filtro Identidad (Para comprobar si funciona correctamente el acceso a memoria compartida)
            {0.0f, 0.0f, 0.0f},
            {0.0f, 1.0f, 0.0f},
            {0.0f, 0.0f, 0.0f}
        },
        {
            // Filtro Edge Detection
            {0.0f, 1.0f, 0.0f},
            {1.0f, -4.0f, 1.0f},
            {0.0f, 1.0f, 0.0f}
        },
        {
            // Filtro Sharpen
            {0.0f, -1.0f, 0.0f},
            {-1.0f, 5.0f, -1.0f},
            {0.0f, -1.0f, 0.0f}
        },
        {
            // Filtro Blur
            {1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f},
            {1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f},
            {1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f}
        },
        {
            // Filtro Emboss
            {-2.0f, -1.0f, 0.0f},
            {-1.0f, 1.0f, 1.0f},
            {0.0f, 1.0f, 2.0f}
        },
        {
            // Filtro Outline
            {-1.0f, -1.0f, -1.0f},
            {-1.0f, 8.0f, -1.0f},
            {-1.0f, -1.0f, -1.0f}
        }
    };

    __shared__ float dataShared[3][ANCHO_IMAGEN + 2][3];

    // Columna del pixel
    int columna = threadIdx.x;

    // Fila del pixel
    int fila = blockIdx.x;

    // Indice global del pixel
    int index = fila * width + columna;
    
    dataShared[1][columna][0] = data[index * 3];
    dataShared[1][columna][1] = data[index * 3 + 1];
    dataShared[1][columna][2] = data[index * 3 + 2];

    // Memoria compartida para los píxeles de la fila superior
    int indixFilaSuperior = (fila - 1) * width + columna;
    dataShared[0][columna][0] = data[indixFilaSuperior * 3];
    dataShared[0][columna][1] = data[indixFilaSuperior * 3 + 1];
    dataShared[0][columna][2] = data[indixFilaSuperior * 3 + 2];

    // Memoria compartida para los píxeles de la fila inferior
    int indexFilaInferior = (fila + 1) * width + columna;
    dataShared[2][columna][0] = data[indexFilaInferior * 3];
    dataShared[2][columna][1] = data[indexFilaInferior * 3 + 1];
    dataShared[2][columna][2] = data[indexFilaInferior * 3 + 2];

    // Sincronizar hilos para asegurar que todos los datos estén en la memoria compartida
    __syncthreads();

    if (fila < height && columna < width) {
        float r = 0.0f, g = 0.0f, b = 0.0f;

        // Aplicar filtro bilineal
        for (int dy = 0; dy <= 2; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                int x = columna + 1 + dx;

                r += dataShared[dy][x][0] * filters[filter][dy][dx + 1];
                g += dataShared[dy][x][1] * filters[filter][dy][dx + 1];
                b += dataShared[dy][x][2] * filters[filter][dy][dx + 1];
            }
        }

        // Por si salen valores negativos o por encima de 255
        r = (r < 0) ? 0 : ((r > 255) ? 255 : r);
        g = (g < 0) ? 0 : ((g > 255) ? 255 : g);
        b = (b < 0) ? 0 : ((b > 255) ? 255 : b);

        // Escribir píxel en la imagen de salida
        dataOut[index * 3] = r;
        dataOut[index * 3 + 1] = g;
        dataOut[index * 3 + 2] = b;
    }
}


vector<float> increaseSize(vector<float> data, int w, int h)
{
    int oldWidth = w;
    int oldHeight = h;

    // Calcular nuevo ancho y alto con el perímetro adicional
    int newWidth = oldWidth + 2;
    int newHeight = oldHeight + 2;

    // Crear nuevo vector para la nueva imagen con el perímetro adicional y rellenar con 0s
    std::vector<float> newImageData(newWidth * newHeight * 3, 0.0f);

    // Copiar los píxeles originales al centro de la nueva imagen
    for (int i = 0; i < oldHeight; i++)
        for (int j = 0; j < oldWidth; j++)
            for (int k = 0; k < 3; k++)
                newImageData[((i + 1) * newWidth * 3 + (j + 1) * 3) + k] = data[(i * oldWidth * 3 + j * 3) + k];

    return newImageData;
}

vector<float> reduceSize(vector<float> data, int w, int h)
{
    int oldWidth = w + 2;

    // Calcular nuevo ancho y alto con el perímetro adicional
    int newWidth = w;
    int newHeight = h;

    // Crear nuevo vector para la nueva imagen con el perímetro adicional y rellenar con 0s
    std::vector<float> newImageData(newWidth * newHeight * 3, 0.0f);

    // Copiar los píxeles originales al centro de la nueva imagen
    for (int i = 0; i < newHeight; i++)
        for (int j = 0; j < newWidth; j++)
            for (int k = 0; k < 3; k++)
                newImageData[(i * newWidth * 3 + j * 3) + k] = data[((i + 1) * oldWidth * 3 + (j + 1) * 3) + k];

    return newImageData;
}



BMPReader::BMPReader(string fileName)
{
    // Open file
    ifstream fIn(fileName);
    char *byteFilePointer = NULL;

    // Read bmp header
    std::vector<char> *bytes = new std::vector<char>(std::istreambuf_iterator<char>(fIn), {});

    byteFilePointer = bytes->data();
    bmpHeader = ((bmpHeader_t *)(byteFilePointer))[0];

    byteFilePointer += sizeof(bmpHeader_t);

    uint32_t bmpType = ((uint32_t *)(byteFilePointer))[0];

    dibHeader.bmpType = bmpType;

    std::cout << "El tipo de bmp es: " << bmpType << endl;

    switch (bmpType)
    {
    case 12:
    {
        dibHeader.dibHeader12 = ((dibHeader12_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader12.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader12.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader12.width * dibHeader.dibHeader12.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    }
    case 40:
    {
        dibHeader.dibHeader40 = ((dibHeader40_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader40.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader40.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader40.width * dibHeader.dibHeader40.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    }
    case 64:
        dibHeader.dibHeader64 = ((dibHeader64_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader64.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader64.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader64.width * dibHeader.dibHeader64.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    case 16:
        dibHeader.dibHeader16 = ((dibHeader16_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader16.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader16.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader16.width * dibHeader.dibHeader16.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    case 52:
        dibHeader.dibHeader52 = ((dibHeader52_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader52.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader52.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader52.width * dibHeader.dibHeader52.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    case 56:
        dibHeader.dibHeader56 = ((dibHeader56_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader56.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader56.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader56.width * dibHeader.dibHeader56.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    case 108:
        dibHeader.dibHeader108 = ((dibHeader108_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader108.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader108.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader108.width * dibHeader.dibHeader108.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    case 124:
        dibHeader.dibHeader124 = ((dibHeader124_t *)(byteFilePointer))[0];
        byteFilePointer = &(bytes->data()[bmpHeader.dataOffset]);
        if (dibHeader.dibHeader124.bpp != 24)
        {
            std::cout << "El formato bpp no soportado: " << dibHeader.dibHeader124.bpp << endl;
        }
        else
        {
            int numPixels = dibHeader.dibHeader124.width * dibHeader.dibHeader124.height;
            rawData.resize(numPixels);
            memcpy(rawData.data(), byteFilePointer, sizeof(pixel24b_t) * numPixels);
            std::cout << "Fichero cargado" << endl;
        }
        break;
    default:
        cerr << "Error: Tipo de encabezado DIB no soportado" << endl;
        break;
    }

    delete bytes;
    fIn.close();
}

vector<float> BMPReader::getRAWData()
{
    std::vector<float> data(rawData.size() * 3);
    for (int i = 0; i < rawData.size(); i++)
    {
        data[i * 3] = rawData[i].r;
        data[i * 3 + 1] = rawData[i].g;
        data[i * 3 + 2] = rawData[i].b;
    }
    return data;
}

void BMPReader::setRAWData(vector<float> data)
{
    rawData.resize(data.size() / 3);
    for (int i = 0; i < rawData.size(); i++)
    {
        rawData[i].r = data[i * 3];
        rawData[i].g = data[i * 3 + 1];
        rawData[i].b = data[i * 3 + 2];
    }
}

void BMPReader::saveBMP(string fileNameOut)
{
    switch (dibHeader.bmpType)
    {
    case 12:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader12, sizeof(dibHeader12_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 40:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader40, sizeof(dibHeader40_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 64:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader64, sizeof(dibHeader64_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 16:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader16, sizeof(dibHeader16_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 52:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader52, sizeof(dibHeader52_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 56:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader56, sizeof(dibHeader56_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 108:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader108, sizeof(dibHeader108_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    case 124:
    {
        ofstream fOut(fileNameOut, ios::binary);
        fOut.write((char *)&bmpHeader, sizeof(bmpHeader_t));
        fOut.write((char *)&dibHeader.dibHeader124, sizeof(dibHeader124_t));
        fOut.write((char *)rawData.data(), rawData.size() * sizeof(pixel24b_t));
        fOut.close();
        break;
    }
    default:
        cerr << "Error: Tipo de encabezado DIB no soportado" << endl;
        break;
    }
}

uint64_t BMPReader::getWidth()
{
    switch (dibHeader.bmpType)
    {
    case 12:
        return dibHeader.dibHeader12.width;
    case 40:
        return dibHeader.dibHeader40.width;
    case 64:
        return dibHeader.dibHeader64.width;
    case 16:
        return dibHeader.dibHeader16.width;
    case 52:
        return dibHeader.dibHeader52.width;
    case 56:
        return dibHeader.dibHeader56.width;
    case 108:
        return dibHeader.dibHeader108.width;
    case 124:
        return dibHeader.dibHeader124.width;
    default:
        cerr << "Error: Tipo de encabezado DIB no soportado" << endl;
        return 0;
    }
}

uint64_t BMPReader::getHeight()
{
    switch (dibHeader.bmpType)
    {
    case 12:
        return dibHeader.dibHeader12.height;
    case 40:
        return dibHeader.dibHeader40.height;
    case 64:
        return dibHeader.dibHeader64.height;
    case 16:
        return dibHeader.dibHeader16.height;
    case 52:
        return dibHeader.dibHeader52.height;
    case 56:
        return dibHeader.dibHeader56.height;
    case 108:
        return dibHeader.dibHeader108.height;
    case 124:
        return dibHeader.dibHeader124.height;
    default:
        cerr << "Error: Tipo de encabezado DIB no soportado" << endl;
        return 0;
    }
}

uint64_t BMPReader::getBPP()
{
    switch (dibHeader.bmpType)
    {
    case 12:
        return dibHeader.dibHeader12.bpp;
    case 40:
        return dibHeader.dibHeader40.bpp;
    case 64:
        return dibHeader.dibHeader64.bpp;
    case 16:
        return dibHeader.dibHeader16.bpp;
    case 52:
        return dibHeader.dibHeader52.bpp;
    case 56:
        return dibHeader.dibHeader56.bpp;
    case 108:
        return dibHeader.dibHeader108.bpp;
    case 124:
        return dibHeader.dibHeader124.bpp;
    default:
        cerr << "Error: Tipo de encabezado DIB no soportado" << endl;
        return 0;
    }
}

void BMPReader::applyBilinearFilter(int filter)
{
    gpuDataManager<float>* dataRes = new gpuDataManager<float>((getHeight() + 2) * (getWidth() + 2) * 3);
    
    // Añadir perímetro
    vector<float> dataIncreased = increaseSize(getRAWData(), getWidth(), getHeight());
    gpuDataManager<float>* dataToFilter = new gpuDataManager<float>(dataIncreased.size());
    for(int i = 0; i < dataIncreased.size(); i++){
        dataToFilter->cpuData[i] = dataIncreased[i];
    }
    dataToFilter->copyToGPU();

    // Calcular bloques/threads
    int numThreadPerBlock = ANCHO_IMAGEN + 2;
    int numBlock = getHeight() + 2;

    cout << "Applying bilinear filter in CUDA" << endl;

    applyBilinearFilter_k<<<numBlock, numThreadPerBlock>>>(getWidth()+2, getHeight()+2, filter, dataToFilter->gpuData, dataRes->gpuData);

    cout << "Waiting CUDA to finish" << endl;

    // Recuperar resultados de GPU
    dataRes->copyToCPU();
    hipDeviceSynchronize();

    cout << "Bilinear filter applied in CUDA" << endl;

    vector<float> dataResCPU(dataRes->size);
    for(int i = 0; i < dataResCPU.size(); i++){
        dataResCPU[i] = dataRes->cpuData[i];
    }

    vector<float> dataReduced = reduceSize(dataResCPU, getWidth(), getHeight());
    setRAWData(dataReduced);

    delete dataRes;
    delete dataToFilter;
}